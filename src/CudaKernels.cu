#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>

__device__ void hsv_to_rgb(float h, float s, float v, float& r, float& g, float& b) {
    if (s <= 0.0f) {
        r = g = b = v;
        return;
    }

    h = fmodf(h, 360.0f) / 60.0f;
    int i = static_cast<int>(h);
    float f = h - i;
    float p = v * (1.0f - s);
    float q = v * (1.0f - s * f);
    float t = v * (1.0f - s * (1.0f - f));

    switch (i) {
        case 0: r = v; g = t; b = p; break;
        case 1: r = q; g = v; b = p; break;
        case 2: r = p; g = v; b = t; break;
        case 3: r = p; g = q; b = v; break;
        case 4: r = t; g = p; b = v; break;
        default: r = v; g = p; b = q; break;
    }
}

__global__ void mandelbrotKernel(uint8_t* pixels, int width, int height,
                                float centerX, float centerY,
                                float scale, int maxIterations)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) return;

    int idx = (py * width + px) * 4;

    // Map pixel to complex plane
    float x0 = centerX + (px - width / 2.0f) * (2.0f * scale / width);
    float y0 = centerY + (py - height / 2.0f) * (2.0f * scale / width);

    float x = 0.0f, y = 0.0f;
    int iteration = 0;
    const float bailout = 65536.0f;

    // Main iteration
    while (x * x + y * y <= bailout && iteration < maxIterations) {
        float x_temp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = x_temp;
        iteration++;
    }

    // Smooth coloring
    float smoothIteration = iteration;
    if (iteration < maxIterations) {
        float log_zn = logf(x * x + y * y) / 2.0f;
        float nu = logf(log_zn / logf(2.0f)) / logf(2.0f);
        smoothIteration = iteration + 1 - nu;
    }

    if (iteration == maxIterations) {
        // Inside set - black
        pixels[idx + 0] = 0;
        pixels[idx + 1] = 0;
        pixels[idx + 2] = 0;
        pixels[idx + 3] = 255;
    } else {
        // Enhanced color mapping
        const float colorFactor = 7.0f;  // Increased for more color variation
        float hue = fmodf(smoothIteration * colorFactor, 360.0f);
        
        // Adjust saturation based on iteration count for more depth
        float saturation = 0.8f + 0.2f * (float)iteration / maxIterations;
        
        // Value (brightness) varies with iteration count
        float value = 0.7f + 0.3f * (float)iteration / maxIterations;
        
        float r, g, b;
        hsv_to_rgb(hue, saturation, value, r, g, b);

        // Write RGBA values
        pixels[idx + 0] = static_cast<uint8_t>(r * 255);
        pixels[idx + 1] = static_cast<uint8_t>(g * 255);
        pixels[idx + 2] = static_cast<uint8_t>(b * 255);
        pixels[idx + 3] = 255;
    }
}

extern "C" void computeMandelbrotCUDA(uint8_t* pixels,
                                   int width, int height,
                                   float centerX, float centerY,
                                   float scale, int maxIterations)
{
    // Default thread block size
    const int threadsPerBlockX = 16;
    const int threadsPerBlockY = 16;
    
    dim3 threadsPerBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Allocate device memory
    uint8_t* d_pixels;
    hipMalloc(&d_pixels, width * height * 4);
    
    // Launch kernel
    mandelbrotKernel<<<numBlocks, threadsPerBlock>>>(d_pixels, width, height,
                                                    centerX, centerY,
                                                    scale, maxIterations);
    
    // Copy result back to host
    hipMemcpy(pixels, d_pixels, width * height * 4, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_pixels);
    
    // Check for errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    }
}
